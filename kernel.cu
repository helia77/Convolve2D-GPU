#include "hip/hip_runtime.h"
﻿#include<iostream>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include<string>
#include<vector>
#include<fstream>

# define PI           3.14159265358979323846  /* pi */

static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "in" << file << "at line" << line;
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 

// ------------------------------------------------------------------------------------------- //

// convolution along x on device
__global__ void dev_conv_x(char* out, char* img, float* kernel, int img_w, int out_h, int out_w, int K) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;	// calculate row index, point to the output
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;	// calculate column index, point to the output
    if (i >= out_h || j >= out_w) return;
    // initialize the register c
    float c[3];
    for (int x = 0; x < 3; x++)
        c[x] = 0.0f;
    
    // apply the convolution with Gaussian kernel along x axis
    for (int k = 0; k < K; k++) {
        c[0] += (unsigned char)img[3 * (i * img_w + j + k)] * kernel[k];
        c[1] += (unsigned char)img[3 * (i * img_w + j + k) + 1] * kernel[k];
        c[2] += (unsigned char)img[3 * (i * img_w + j + k) + 2] * kernel[k];
    }
    out[3 * (i * out_w + j)] = c[0];
    out[3 * (i * out_w + j) + 1] = c[1];
    out[3 * (i * out_w + j) + 2] = c[2];
}


__global__ void intensity(char* out, char* img, int img_w, int img_h) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;	// calculate row index, point to the output
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;	// calculate column index, point to the output
    if (i >= img_h || j >= img_w) return;
    out[3 * (i * img_w + j)] = 0.5 * (unsigned char)img[3 * (i * img_w + j)];
    out[3 * (i * img_w + j) + 1] = 0.5 * (unsigned char)img[3 * (i * img_w + j) + 1];
    out[3 * (i * img_w + j) + 2] = 0.5 * (unsigned char)img[3 * (i * img_w + j) + 2];
}




//  convolution along y on device
__global__ void dev_conv_y(char* out, char* img, float* kernel, int img_w, int out_h, int out_w, int K) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= out_h || j >= out_w) return;
    float c[3];
    for (int x{}; x < 3; x++)
        c[x] = 0.0f;
    for (int k{}; k < K; k++) {
        c[0] += (unsigned char)img[3 * ((i + k) * img_w + j)] * kernel[k];
        c[1] += (unsigned char)img[3 * ((i + k) * img_w + j) + 1] * kernel[k];
        c[2] += (unsigned char)img[3 * ((i + k) * img_w + j) + 2] * kernel[k];
    }
    out[3 * (i * out_w + j)] = c[0];
    out[3 * (i * out_w + j) + 1] = c[1];
    out[3 * (i * out_w + j) + 2] = c[2];
}


// ------------------------------------------------------------------------------------------- //


// convolution kernel along x running on host
void host_conv_x(char* out, char* img, float* kernel, int img_w, int out_h, int out_w, int K) {
    float c[3];
    for (int i = 0; i < out_h; i++) {										// calculate the i (row) index, point to the outMatrix
        for (int j = 0; j < out_w; j++) {									// calculate the j (column) index, point to the outMatrix
            // initialize the register c to store the results
            for (int x{}; x < 3; x++)
                c[x] = 0.0f;
            // convolving with Gaussian kernel along x axis
            for (int k{}; k < K; k++) {
                c[0] += (unsigned char)img[3 * (i * img_w + j + k)] * kernel[k];
                c[1] += (unsigned char)img[3 * (i * img_w + j + k) + 1] * kernel[k];
                c[2] += (unsigned char)img[3 * (i * img_w + j + k) + 2] * kernel[k];
            }
            out[3 * (i * out_w + j)] = c[0];
            out[3 * (i * out_w + j) + 1] = c[1];
            out[3 * (i * out_w + j) + 2] = c[2];
        }
    }
}

// convolution kernel along y running on host, same as convolution_x
void host_conv_y(char* out, char* img, float* kernel, int inp_w, int out_h, int out_w, int K) {
    float c[3];
    for (int j = 0; j < out_w; j++) {
        for (int i = 0; i < out_h; i++) {
            for (int x = 0; x < 3; x++)
                c[x] = 0.0f;
            for (int k = 0; k < K; k++) {
                c[0] += (unsigned char)img[3 * ((i + k) * inp_w + j)] * kernel[k];
                c[1] += (unsigned char)img[3 * ((i + k) * inp_w + j) + 1] * kernel[k];
                c[2] += (unsigned char)img[3 * ((i + k) * inp_w + j) + 2] * kernel[k];
            }
            out[3 * (i * out_w + j)] = c[0];
            out[3 * (i * out_w + j) + 1] = c[1];
            out[3 * (i * out_w + j) + 2] = c[2];
        }
    }
}


// ------------------------------------------------------------------------------------------- //


void write_tga(std::string filename, char* bytes, int width, int height) {
    std::ofstream outfile;
    outfile.open(filename, std::ios::binary | std::ios::out);	// open a binary file
    outfile.put(0);						// id length (field 1)
    outfile.put(0);						// color map type (field 2)
    outfile.put(2);						// image_type (field 3)
    outfile.put(0); outfile.put(0);		// color map field entry index (field 4)
    outfile.put(0); outfile.put(0);		// color map length (field 4)
    outfile.put(0);				// color map entry size (field 4)
    outfile.put(0); outfile.put(0);		// x origin (field 5)
    outfile.put(0); outfile.put(0);		// y origin (field 5)
    outfile.write((char*)&width, 2);		// image width (field 5)
    outfile.write((char*)&height, 2);		// image height (field 5)
    outfile.put(24);				// pixel depth (field 5)
    outfile.put(0);				// image descriptor (field 5)
    outfile.write(bytes, width * height * 3);		// write the image data
    outfile.close();				// close the file
}

std::vector<char> read_tga(std::string filename, int& width, int& height) {
    std::ifstream infile;
    infile.open(filename, std::ios::binary | std::ios::out);        // open the file for binary writing
    if (!infile.is_open()) {
        std::cout << "ERROR: Unable to open file " << filename << std::endl;
        return std::vector<char>();
    }
    char id_length;                                infile.get(id_length);                            // id length (field 1)
    char cmap_type;                                infile.get(cmap_type);                            // color map type (field 2)
    char image_type;                            infile.get(image_type);                        // image_type (field 3)
    char field_entry_a, field_entry_b;
    infile.get(field_entry_a);                infile.get(field_entry_b);                        // color map field entry index (field 4)
    char map_length_a, map_length_b;
    infile.get(map_length_a);                infile.get(map_length_b);                        // color map field entry index (field 4)
    char map_size;                                infile.get(map_size);                            // color map entry size (field 4)
    char origin_x_a, origin_x_b;
    infile.get(origin_x_a);                infile.get(origin_x_b);                        // x origin (field 5)
    char origin_y_a, origin_y_b;
    infile.get(origin_y_a);                infile.get(origin_y_b);                        // x origin (field 5)

    infile.read((char*)&width, 2);
    infile.read((char*)&height, 2);
    char pixel_depth;                            infile.get(pixel_depth);
    char descriptor;                            infile.get(descriptor);

    std::vector<char> bytes(width * height * 3);
    infile.read(&bytes[0], width * height * 3);

    infile.close();                    // close the file

    return bytes;
}

int main(int argc, char* argv[]) {
    //std::cout << "Convolve image.tga 40";
    if (argc != 3) {
        fprintf(stderr, "Error: 3 parameters expected. Found %d\n", argc);
        return 1;
    }

    std::string filename(argv[1]);
    int sigma = atoi(argv[2]);

    std::cout << "Filename: " << filename << std::endl;
    std::cout << "Input Sigma: " << sigma << std::endl;
    
    //Calculating kernel size
    int k_size = 4 * sigma;                          //radius = 2 * sigma
                                                     //kernel size = 2 * radius + 1
    if (k_size % 2 == 0) k_size++;                   //kernel size should be odd
    float miu = k_size / 2;

    int width = 0;		
    int height = 0;		
    int size = 0;		// size of the image array
    // No padding
    // strides = 1

    // loading image into a char* array
    std::cout << "Loading the image..." << std::endl;
    std::vector<char> imageVector = read_tga(filename, width, height);
    std::cout << "Loading finished.\n" <<  "width: " << width << " height: " << height << std::endl;
    size = width * height * 3;
    char* imageArray = &imageVector[0];

    // array sizes after convolution along x axis
    int x_height = height;
    int x_width = width - k_size + 1;
    int x_size = x_height * x_width * 3;
    char* x_output = (char*)malloc(x_size * sizeof(char));
    std::cout << "Part 1 finished." << std::endl;

    // array sizes after convolution along y axis
    int y_height = x_height - k_size + 1;
    int y_width = x_width;
    int y_size = y_height * y_width * 3;
    char* y_output = (char*)malloc(y_size * sizeof(char));
    std::cout << "Part 2 finished." << std::endl;

    // gaussian kernel as a float*
    float* gKernel = (float*)malloc(k_size * sizeof(float));
    int s = 2 * sigma * sigma;
    for (int i = 0; i < k_size; i++) {
        gKernel[i] = 1 / sqrt(s * (float)PI) * exp(-(i - miu)*(i - miu) / s);
        //std::cout << gKernel[i] << " | ";
        //return 0;
    }



    // running on host or device starts
    int device{};
    std::cout << "Host(1) or device(0)?" << std::endl;
    std::cin >> device;

    if (device) {
        // -------------------------------------- CPU ---------------------------------------- //

        std::cout << "------------------------- CPU version -------------------------" << std::endl;
        std::cout << "Convolving on HOST..." << std::endl;

        clock_t start, finish;
        start = clock();

        // convolving along x
        host_conv_x(x_output, imageArray, gKernel, width, x_height, x_width, k_size);
        // convolving along y 
        host_conv_y(y_output, x_output, gKernel, x_width, y_height, y_width, k_size);

        finish = clock();  // time finishs
        std::cout << "It takes " << (double)(finish - start) / CLOCKS_PER_SEC << " s to convolve on CPU" << std::endl;
        write_tga("out.tga", y_output, y_width, y_height);
        std::cout << "Convolution on CPU finished" << std::endl;
        //// -------------------------------------- CPU ---------------------------------------- //
    }


    else {
        //// -------------------------------------- GPU ---------------------------------------- //

        std::cout << "------------------------- GPU version -------------------------" << std::endl;
        int d;
        HANDLE_ERROR(hipGetDevice(&d));
        std::cout << "Current device: " << d << std::endl;
        hipDeviceProp_t prop;
        HANDLE_ERROR(hipGetDeviceProperties(&prop, d));
        
        float* gpu_gKernel;
        char* gpu_image;
        char* gpu_output_x;
        char* gpu_output_y;
        char* gpu_image_i;
        std::cout << "Image size: " << size << std::endl;

        // allocate memory for image, kernel, and two convoled outputs
        //HANDLE_ERROR(hipMalloc(&gpu_gKernel, k_size * sizeof(float)));
        HANDLE_ERROR(hipMalloc(&gpu_image, size * sizeof(char)));
        //HANDLE_ERROR(hipMalloc(&gpu_output_x, x_size * sizeof(char)));
        //HANDLE_ERROR(hipMalloc(&gpu_output_y, y_size * sizeof(char))); 
        HANDLE_ERROR(hipMalloc(&gpu_image_i, size * sizeof(char)));
        // copy image and kernel from main memory to Device
        HANDLE_ERROR(hipMemcpy(gpu_image, imageArray, size * sizeof(char), hipMemcpyHostToDevice));
        //HANDLE_ERROR(hipMemcpy(gpu_gKernel, gKernel, k_size * sizeof(float), hipMemcpyHostToDevice));
        
        size_t blockDim = sqrt(prop.maxThreadsPerBlock);
        dim3 threads(blockDim, blockDim);
        std::cout << "threads.x: " << threads.x << ",\tthreads.y: " << threads.y << std::endl;
        dim3 blocks(width / threads.x + 1, height / threads.y + 1);
        char* imgoutput = (char*)malloc(size * sizeof(char));

        // starting GPU timer
        hipEvent_t g_start;
        hipEvent_t g_stop;
        hipEventCreate(&g_start);
        hipEventCreate(&g_stop);
        hipEventRecord(g_start, NULL);


        std::cout << "Convolving on DEVICE..." << std::endl;
        // convolving along x
        //dev_conv_x <<< blocks, threads >>> (gpu_output_x, gpu_image, gpu_gKernel, width, x_height, x_width, k_size);
        // convolving along y
        //dev_conv_y <<< blocks, threads >>> (gpu_output_y, gpu_output_x, gpu_gKernel, x_width, y_height, y_width, k_size);

        intensity << < blocks, threads >> > (gpu_image_i, gpu_image, width, height);

        // copy convolved outputs from Device to main memory
        //HANDLE_ERROR(hipMemcpy(x_output, gpu_output_x, x_size * sizeof(char), hipMemcpyDeviceToHost));
        //HANDLE_ERROR(hipMemcpy(y_output, gpu_output_y, y_size * sizeof(char), hipMemcpyDeviceToHost));
        HANDLE_ERROR(hipMemcpy(imgoutput, gpu_image_i, size * sizeof(char), hipMemcpyDeviceToHost));
        // timer ends
        hipEventRecord(g_stop, NULL);
        hipEventSynchronize(g_stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, g_start, g_stop);
        std::cout << "It takes " << elapsedTime << " ms to convolve on GPU" << std::endl;

        // output file
        //write_tga("out_GPU_x.tga", x_output, x_width, x_height);
        write_tga("out_GPU.tga", imgoutput, width, height);

        std::cout << "Convolution on GPU finished" << std::endl;

        //// -------------------------------------- GPU ---------------------------------------- //
    }
}