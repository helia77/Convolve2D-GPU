#include "hip/hip_runtime.h"
﻿#include<iostream>
#include "hip/hip_runtime.h"
#include ""
#include<string>
#include<vector>
#include<fstream>

# define PI           3.14159265358979323846  /* pi */

static void HandleError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << "in" << file << "at line" << line;
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 

// ------------------------------------------------------------------------------------------- //


// convolution kernel along x running on device
__global__ void dev_convolve_x(char* C, char* A, float* B, int M, int N, int M_x, int N_x, int K) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;	// calculate the i (row) index, point to the outMatrix
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;	// calculate the j (column) index, point to the outMatrix
    if (i >= M_x || j >= N_x) return;
    // initialize the register c to store the results
    float c[3];
    for (int x = 0; x < 3; x++)
        c[x] = 0;
    //kernelConvolution_x_ns<<<blocks, threads>>>(gpu_outArray_x, gpu_inArray, gpu_gKernel, height, width, height_x, width_x, k_size);
    // apply the convolution with Gaussian kernel along x axis
    for (int k = 0; k < K; k++) {
        c[0] += (float)A[3 * (i * N + j + k)] * B[k];
        c[1] += (float)A[3 * (i * N + j + k) + 1] * B[k];
        c[2] += (float)A[3 * (i * N + j + k) + 2] * B[k];
    }
    // copy results from register to outMatrix
    C[3 * (i * N_x + j)] = (char)c[0];
    C[3 * (i * N_x + j) + 1] = (char)c[1];
    C[3 * (i * N_x + j) + 2] = (char)c[2];
}

// convolution kernel along y running on device
__global__ void dev_convolve_y(char* C, char* A, float* B, int M, int N, int M_y, int N_y, int K) {
    size_t i = blockDim.y * blockIdx.y + threadIdx.y;
    size_t j = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= M_y || j >= N_y) return;
    float c[3];
    for (int x = 0; x < 3; x++)
        c[x] = 0;
    for (int k = 0; k < K; k++) {
        c[0] += (float)A[3 * ((i + k) * N + j)] * B[k];
        c[1] += (float)A[3 * ((i + k) * N + j) + 1] * B[k];
        c[2] += (float)A[3 * ((i + k) * N + j) + 2] * B[k];
    }
    C[3 * (i * N_y + j)] = (char)c[0];
    C[3 * (i * N_y + j) + 1] = (char)c[1];
    C[3 * (i * N_y + j) + 2] = (char)c[2];
}


// ------------------------------------------------------------------------------------------- //


// convolution kernel along x running on host
void host_convolve_x(char* out, char* img, float* gaus, int img_w, int out_h, int out_w, int K) {
    float c[3];
    for (int i = 0; i < out_h; i++) {										// calculate the i (row) index, point to the outMatrix
        for (int j = 0; j < out_w; j++) {									// calculate the j (column) index, point to the outMatrix
            // initialize the register c to store the results
            for (int x = 0; x < 3; x++)     c[x] = 0;
            // apply the convolution with Gaussian kernel along x axis
            for (int k = 0; k < K; k++) {
                c[0] += (float)img[3 * (i * img_w + j + k)] * gaus[k];
                c[1] += (float)img[3 * (i * img_w + j + k) + 1] * gaus[k];
                c[2] += (float)img[3 * (i * img_w + j + k) + 2] * gaus[k];
            }
            // copy results from register to outMatrix
            out[3 * (i * out_w + j)] = (char)c[0];
            out[3 * (i * out_w + j) + 1] = (char)c[1];
            out[3 * (i * out_w + j) + 2] = (char)c[2];
        }
    }
}

// convolution kernel along y running on host, same as convolution_x
void host_convolve_y(char* out, char* img, float* gaus, int inp_w, int out_h, int out_w, int K) {
    float c[3];
    for (int j = 0; j < out_w; j++) {
        for (int i = 0; i < out_h; i++) {
            for (int x = 0; x < 3; x++)     c[x] = 0;

            for (int k = 0; k < K; k++) {
                c[0] += (float)img[3 * ((i + k) * inp_w + j)] * gaus[k];
                c[1] += (float)img[3 * ((i + k) * inp_w + j) + 1] * gaus[k];
                c[2] += (float)img[3 * ((i + k) * inp_w + j) + 2] * gaus[k];
            }
            out[3 * (i * out_w + j)] = (char)c[0];
            out[3 * (i * out_w + j) + 1] = (char)c[1];
            out[3 * (i * out_w + j) + 2] = (char)c[2];
        }
    }
}


// ------------------------------------------------------------------------------------------- //


void write_tga(std::string filename, char* bytes, short width, short height) {
    std::ofstream outfile;
    outfile.open(filename, std::ios::binary | std::ios::out);	// open a binary file
    outfile.put(0);						// id length (field 1)
    outfile.put(0);						// color map type (field 2)
    outfile.put(2);						// image_type (field 3)
    outfile.put(0); outfile.put(0);		// color map field entry index (field 4)
    outfile.put(0); outfile.put(0);		// color map length (field 4)
    outfile.put(0);				// color map entry size (field 4)
    outfile.put(0); outfile.put(0);		// x origin (field 5)
    outfile.put(0); outfile.put(0);		// y origin (field 5)
    outfile.write((char*)&width, 2);		// image width (field 5)
    outfile.write((char*)&height, 2);		// image height (field 5)
    outfile.put(24);				// pixel depth (field 5)
    outfile.put(0);				// image descriptor (field 5)
    outfile.write(bytes, width * height * 3);		// write the image data
    outfile.close();				// close the file
}

std::vector<char> read_tga(std::string filename, short& width, short& height) {
    std::ifstream infile;
    infile.open(filename, std::ios::binary | std::ios::out);        // open the file for binary writing
    if (!infile.is_open()) {
        std::cout << "ERROR: Unable to open file " << filename << std::endl;
        return std::vector<char>();
    }
    char id_length;                                infile.get(id_length);                            // id length (field 1)
    char cmap_type;                                infile.get(cmap_type);                            // color map type (field 2)
    char image_type;                            infile.get(image_type);                        // image_type (field 3)
    char field_entry_a, field_entry_b;
    infile.get(field_entry_a);                infile.get(field_entry_b);                        // color map field entry index (field 4)
    char map_length_a, map_length_b;
    infile.get(map_length_a);                infile.get(map_length_b);                        // color map field entry index (field 4)
    char map_size;                                infile.get(map_size);                            // color map entry size (field 4)
    char origin_x_a, origin_x_b;
    infile.get(origin_x_a);                infile.get(origin_x_b);                        // x origin (field 5)
    char origin_y_a, origin_y_b;
    infile.get(origin_y_a);                infile.get(origin_y_b);                        // x origin (field 5)

    infile.read((char*)&width, 2);
    infile.read((char*)&height, 2);
    char pixel_depth;                            infile.get(pixel_depth);
    char descriptor;                            infile.get(descriptor);

    std::vector<char> bytes(width * height * 3);
    infile.read(&bytes[0], width * height * 3);

    infile.close();                    // close the file

    return bytes;
}

int main(int argc, char* argv[]) {
    //std::cout << "Convolve image.tga 40";
    if (argc != 3) {
        fprintf(stderr, "Error: 3 parameters expected. Found %d\n", argc);
        return 1;
    }

    std::string filename(argv[1]);
    int sigma = atoi(argv[2]);

    std::cout << "filename: " << filename << std::endl;
    std::cout << "sigma: " << sigma << std::endl;
    
    int k_size = 4 * sigma;
    if (k_size % 2 == 0) k_size++; //make sure k is odd
    float miu = k_size / 2;

    //char pSix[10];		// indicates this is a PPM image
    short width = 0;		// width of the image
    short height = 0;		// height of the image
    int maximum = 0;	// maximum pixel value
    int size = 0;		// size of the array


    // read the PPM file and store its contents inside an array and return the pointer to that array to pixelArray
    // notice the type of the array is unsigned char!!!!
    std::cout << "Starts loading." << std::endl;
    std::vector<char> imageVector = read_tga(filename, width, height);
    std::cout << "Load finished." << std::endl;
    std::cout << "width: " << width << " height: " << height << std::endl;
    size = width * height * 3;
    char* imageArray = &imageVector[0];

    // allocate output array for pixels after convolution along x axis
    int height_x = height;
    int width_x = (width - k_size) + 1;
    int size_x = height_x * width_x * 3;
    char* outArray_x = (char*)malloc(size_x * sizeof(char));
    std::cout << "Part 1 finished." << std::endl;
    // allocate output array for pixels after convolution along y axis
    int height_y = (height_x - k_size) + 1;
    int width_y = width_x;
    int size_y = height_y * width_y * 3;
    char* outArray_y = (char*)malloc(size_y * sizeof(char));
    std::cout << "Part 2 finished." << std::endl;
    // define a float pointer to the gaussian kernel
    float* gKernel = (float*)malloc(k_size * sizeof(float));
    int s = 2 * sigma * sigma;
    int sum{5};

    for (int i = 0; i < k_size; i++) {
        gKernel[i] = 1 / sqrt(s * (float)PI) * exp(-(i - miu+1)*(i - miu+1) / s);
        //std::cout << gKernel[i] << " | ";
        sum = sum + gKernel[i];
        //return 0;
    }
    //return 0;
    int device{};
    std::cout << "Host(1) or device(0)?" << std::endl;
    std::cin >> device;

    if (device) {
        // -------------------------------------- CPU VERSION ---------------------------------------- //

        std::cout << "--------------------- CPU version ---------------------" << std::endl;
        std::cout << "Starts doing convolution on CPU" << std::endl;

        clock_t start, finish;	//Create timer
        double duration;
        start = clock();  // time starts

        // do kernel convolution along x axis
        host_convolve_x(outArray_x, imageArray, gKernel, width, height_x, width_x, k_size);
        // do kernel convolution along y axis
        host_convolve_y(outArray_y, outArray_x, gKernel, width_x, height_y, width_y, k_size);

        finish = clock();  // time ends
        duration = (double)(finish - start) / CLOCKS_PER_SEC;
        std::cout << "It takes " << duration << " s to do the CPU based convolution!" << std::endl;
        write_tga("outy.tga", outArray_y, width_y, height_y);
        std::cout << "Convolution on CPU finished" << std::endl;
        //// -------------------------------------- CPU VERSION ---------------------------------------- //
    }
    else {
        //// -------------------------------------- GPU VERSION ---------------------------------------- //

        std::cout << "--------------------- GPU version ---------------------" << std::endl;
        hipDeviceProp_t props;																//declare a CUDA properties structure
        HANDLE_ERROR(hipGetDeviceProperties(&props, 0));									//get the properties of the first CUDA device

        float* gpu_gKernel;																	//pointer to the gaussian kernel
        char* gpu_inArray;  																//pointer to input array (image)
        char* gpu_outArray_x;  																//pointer to output Array after convolution along x
        char* gpu_outArray_y;  																//pointer to output Array after all the convolution

        std::cout << "size: " << size << std::endl;

        HANDLE_ERROR(hipMalloc(&gpu_gKernel, k_size * sizeof(float)));  							//allocate memory on device
        HANDLE_ERROR(hipMalloc(&gpu_inArray, size * sizeof(char)));  							    //allocate memory on device
        HANDLE_ERROR(hipMalloc(&gpu_outArray_x, size_x * sizeof(char)));  							//allocate memory on device
        HANDLE_ERROR(hipMalloc(&gpu_outArray_y, size_y * sizeof(char)));  							//allocate memory on device

        HANDLE_ERROR(hipMemcpy(gpu_gKernel, gKernel, k_size * sizeof(float), hipMemcpyHostToDevice));  //copy the array from main memory to device
        HANDLE_ERROR(hipMemcpy(gpu_inArray, imageArray, size * sizeof(char), hipMemcpyHostToDevice));     //copy the array from main memory to device

        size_t blockDim = sqrt(props.maxThreadsPerBlock);
        dim3 threads(blockDim, blockDim);
        std::cout << "threads.x: " << threads.x << std::endl;
        std::cout << "threads.y: " << threads.y << std::endl;
        dim3 blocks(width / threads.x + 1, height / threads.y + 1);


        // without shared memory
        std::cout << "Convolving on GPU..." << std::endl;

        // utilize hipEvent_t to serve as GPU timer
        hipEvent_t d_start;
        hipEvent_t d_stop;
        hipEventCreate(&d_start);
        hipEventCreate(&d_stop);
        hipEventRecord(d_start, NULL);

        dev_convolve_x << <blocks, threads >> > (gpu_outArray_x, gpu_inArray, gpu_gKernel, height, width, height_x, width_x, k_size);
        dev_convolve_y << <blocks, threads >> > (gpu_outArray_y, gpu_outArray_x, gpu_gKernel, height_x, width_x, height_y, width_y, k_size);

        HANDLE_ERROR(hipMemcpy(outArray_x, gpu_outArray_x, size_x * sizeof(char), hipMemcpyDeviceToHost));  //copy the array back from device to main memory
        HANDLE_ERROR(hipMemcpy(outArray_y, gpu_outArray_y, size_y * sizeof(char), hipMemcpyDeviceToHost));  //copy the array back from device to main memory

        //	end of hipEvent_t, calculate the time and show
        hipEventRecord(d_stop, NULL);
        hipEventSynchronize(d_stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, d_start, d_stop);
        std::cout << "It takes " << elapsedTime << " ms to do the GPU based convolution!" << std::endl;

        // output file
        //write_tga("out_GPU_x.tga", outArray_x, width_x, height_x);
        write_tga("out_GPU_y.tga", outArray_y, width_y, height_y);

        std::cout << "Convolution on GPU finished" << std::endl;
    }
}